#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include<time.h>
#define blknum 128
#define N 10000000
__global__ void add( int *a, int *b, int *c ){
    int tid = threadIdx.x + blockIdx.x*blockDim.x;
	int stride = blockDim.x*blockDim.y*gridDim.x*gridDim.y;
	for(int i=tid;i<N;i+=stride){
		//printf("%d\n",i);
		c[i] = a[i] + b[i];
	}
//	if(blkid<N)
//		c[blkid] = a[blkid]+b[blkid];
}


int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
	
    // allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );
	
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
	srand ( time(NULL) );
    for (int i=0; i<N; i++) {
        a[i] = rand()%256;
        b[i] = rand()%256;
    }
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
   
	/* Get start time event{{{*/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	/*}}}*/

	//GPU kernel function
    add<<<blknum,1>>>( dev_a, dev_b, dev_c ); //1 block 1024 thread
    
	/* Get stop time event{{{*/
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); 
	// Compute execution time
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU time: %13f msec\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	/*}}}*/
    
    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}
