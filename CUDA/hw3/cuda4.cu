#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include<time.h>
#define blknum 1
#define threadnum 1024
#define N 10000000
__global__ void add( int *a, int *b, int *c ){
    int tid = threadIdx.y* blockDim.x + threadIdx.x;
	int blkid = blockDim.x;
	for(int i=tid;i<N;i+=blkid){
		//printf("%d\n",i);
		c[i] = a[i] + b[i];
	}
//	if(blkid<N)
//		c[blkid] = a[blkid]+b[blkid];
}


int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
	
    // allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );
	
    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    // fill the arrays 'a' and 'b' on the CPU
	srand ( time(NULL) );
    for (int i=0; i<N; i++) {
        a[i] = rand()%256;
        b[i] = rand()%256;
    }
    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy( dev_a, a, N * sizeof(int), hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int), hipMemcpyHostToDevice );
   
	/* Get start time event{{{*/
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	/*}}}*/
	

	
	//GPU kernel function
	   	 
	dim3 blocksize(64,16);
	add<<<blocknum,blocksize>>>( dev_a, dev_b, dev_c ); //1 block 1024 thread
    
	/* Get stop time event{{{*/
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); 
	// Compute execution time
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("GPU time: %13f msec\n", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	/*}}}*/
    
    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy( c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost );
    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // free the memory allocated on the GPU
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}
