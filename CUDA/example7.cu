#include<stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_ROWS 4
#define NUM_COLS 4

 __global__ void add2(int* da, int* db, int* dc)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;
    int id = idy * gridDim.x * blockDim.x  + idx;
    dc[id] = da[id]+ db[id];
}

int main()
{
    int	ha[NUM_ROWS][NUM_COLS] ;
    int	*da ;
    int	hb[NUM_ROWS][NUM_COLS] ;
    int *db;
    int	hc[NUM_ROWS][NUM_COLS] ;
    int *dc;
	
    int iSize = NUM_ROWS * NUM_COLS * sizeof(int) ;
    hipError_t     cuError = hipSuccess;
	
    dim3 dimGrid (NUM_ROWS/2, NUM_COLS/2, 1) ;
    dim3 dimBlock (NUM_ROWS/2, NUM_COLS/2, 1) ;

    for(int i=0;i<NUM_ROWS;i++)
    {
        for(int j=0;j<NUM_COLS;j++)
        {
            ha[i][j]=rand()%10+1;
	    hb[i][j]=rand()%10+1;
        }
    }

    cuError = hipMalloc((void**)&da, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("Failed to allocate memory\n") ;
        return 1 ;
    }
    cuError = hipMemcpy(da, ha, iSize, hipMemcpyHostToDevice);
    if (hipSuccess != cuError)
    {
        hipFree (da) ;
        printf ("Failed in Memcpy 1\n") ;
        return 1 ;
    }
	
    cuError = hipMalloc((void**)&db, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("Failed to allocate memory\n") ;
        return 1 ;
    }
    cuError = hipMemcpy(db, hb, iSize, hipMemcpyHostToDevice);
    if (hipSuccess != cuError)
    {
        hipFree (db) ;
        printf ("Failed in Memcpy 1\n") ;
        return 1 ;
    }
    cuError = hipMalloc((void**)&dc, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("Failed to allocate memory\n") ;
        return 1 ;
    }


    add2<<<dimGrid, dimBlock>>>(da, db, dc);
	
    cuError = hipGetLastError () ;
    if (hipSuccess != cuError)
    {
            printf ("Failed in kernel launch and reason is %s\n", hipGetErrorString(cuError)) ;
            return 1 ;
    }

    cuError = hipMemcpy(hc, dc, iSize, hipMemcpyDeviceToHost);
    if (hipSuccess != cuError)
    {
            hipFree (dc) ;
            printf ("Failed in Memcpy 2\n") ;
            return 1 ;
    }

    bool success = true;
    for(int i=0;i<NUM_ROWS;i++){
        for(int j=0;j<NUM_COLS;j++){
            if ((ha[i][j] + hb[i][j]) != hc[i][j]) {
		 printf( "Error:  %d + %d != %d\n", ha[i][j], hb[i][j], hc[i][j] );
		 success = false;
	    } 
	}
    }
    if (success)    printf( "We did it!\n" );
	
    hipFree (da) ;
    hipFree (db) ;	
    hipFree (dc) ;

    return 0;
}
